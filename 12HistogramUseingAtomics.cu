#include "hip/hip_runtime.h"
// Name:Alaina Odham
// Histogram useing atomics in global memory and shared memory.
// nvcc 12HistogramUseingAtomics.cu -o temp

/*
 What to do:
 This code generates a series of random numbers and places them into bins based on size ranges using the CPU.

 Your task:
 - Create a binning scheme that utilizes the GPU.
 - Take advantage of both global and shared memory atomic operations.
 - The function call has already been provided.
 - Set the block size to **twice** the number of multiprocessors on the GPU.
*/

/*
 Purpose:
 To learn how to use atomic operations at both the shared and global memory levels.
 Along the way, you'll also learn a bit about generating random numbers using `srand`,
 which will come in handy when we use `hiprand` in a later assignment.
*/

// Include files
#include <sys/time.h>
#include <stdio.h>

/*
 Note: The Max int value is 2,147,483,647
 The length of the sequence of random number that srand generates is 2^32,
 that is 4,294,967,296 which is bigger than the largest int but the max for an unsigned int.
*/

// Defines
#define NUMBER_OF_RANDOM_NUMBERS 2147483
#define NUMBER_OF_BINS 10
#define MAX_RANDOM_NUMBER 100.0f

// Global variables
float *RandomNumbersGPU;
int *HistogramGPU;
float *RandomNumbersCPU;
int *HistogramCPU;
int *HistogramCPUTemp; // Use it to hold the GPU histogram past back so we can compare to CPU histogram.
dim3 BlockSize; //This variable will hold the Dimensions of your block
dim3 GridSize; //This variable will hold the Dimensions of your grid

//Function prototypes
void cudaErrorCheck(const char *, int);
void SetUpCudaDevices();
void AllocateMemory();
void Innitialize();
void CleanUp();
void fillHistogramCPU();
__global__ void fillHistogramGPU(float *, int *);
int main();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

//This will be the layout of the parallel space we will be using.
void SetUpCudaDevices()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	cudaErrorCheck(__FILE__, __LINE__);
	
	BlockSize.x = 2 * prop.multiProcessorCount;//we do this because it gives a thread something else to work on if one block is stalling(we only do *2 to not max out shared memory)
	if(prop.maxThreadsDim[0] < BlockSize.x)
	{
		printf("\n You are trying to create more threads (%d) than your GPU can support on a block (%d).\n Good Bye\n", BlockSize.x, prop.maxThreadsDim[0]);
		exit(0);
	}
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = (NUMBER_OF_RANDOM_NUMBERS - 1)/BlockSize.x + 1; //Makes enough blocks to deal with the whole vector.
	if(prop.maxGridSize[0] < GridSize.x)
	{
		printf("\n You are trying to create more blocks (%d) than your GPU can suppport (%d).\n Good Bye\n", GridSize.x, prop.maxGridSize[0]);
		exit(0);
	}
	GridSize.y = 1;
	GridSize.z = 1;
}

//Sets memory on the GPU and CPU for our use.
void AllocateMemory()
{					
	//Allocate Device (GPU) Memory
	hipMalloc(&RandomNumbersGPU, NUMBER_OF_RANDOM_NUMBERS*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&HistogramGPU, NUMBER_OF_BINS*sizeof(int));
	cudaErrorCheck(__FILE__, __LINE__);

	//Allocate Host (CPU) Memory
	RandomNumbersCPU = (float*)malloc(NUMBER_OF_RANDOM_NUMBERS*sizeof(float));
	HistogramCPU = (int*)malloc(NUMBER_OF_BINS*sizeof(int));
	HistogramCPUTemp = (int*)malloc(NUMBER_OF_BINS*sizeof(int));
	
	//Setting the the histograms to zero.
	hipMemset(HistogramGPU, 0, NUMBER_OF_BINS*sizeof(int));
	cudaErrorCheck(__FILE__, __LINE__);
	memset(HistogramCPU, 0, NUMBER_OF_BINS*sizeof(int));
}

//Loading random numbers.
void Innitialize()
{
	time_t t;
	srand((unsigned) time(&t));
	
	// rand() returns an int in [0, RAND_MAX] "end points included".
	
	for(int i = 0; i < NUMBER_OF_RANDOM_NUMBERS; i++)
	{		
		RandomNumbersCPU[i] = MAX_RANDOM_NUMBER*(float)rand()/RAND_MAX;	
	}
}

//Cleaning up memory after we are finished.
void CleanUp()
{
	hipFree(RandomNumbersGPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(HistogramGPU);
	cudaErrorCheck(__FILE__, __LINE__);
	free(RandomNumbersCPU); 
	free(HistogramCPU);
	free(HistogramCPUTemp);
	//printf("\n Cleanup Done.");
}

void fillHistogramCPU()
{
	float breakPoint;
	int k, done;
	float stepSize = MAX_RANDOM_NUMBER/(float)NUMBER_OF_BINS;
	
	for(int i = 0; i < NUMBER_OF_RANDOM_NUMBERS; i++)
	{
		breakPoint = stepSize;
		k = 0;
		done =0;
		while(done == 0)
		{
			if(RandomNumbersCPU[i] < breakPoint)
			{
				HistogramCPU[k]++; 
				done = 1;
			}
			
			if(NUMBER_OF_BINS < k)
			{
				printf("\n k is too big\n");
				exit(0);
			}
			k++;
			breakPoint += stepSize;
		}
	}
}

//This is the kernel. It is the function that will run on the GPU.
__global__ void fillHistogramGPU(float *randomNumbers, int *hist)
{
	// make a shared memory for each block
    __shared__ int sharedHist[NUMBER_OF_BINS];

    // assign enough thread ids for each random number
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

	//initialize shared memory to zero
    for (int i = threadIdx.x; i < NUMBER_OF_BINS; i = i+blockDim.x)
	{
        sharedHist[i] = 0;
	}
	//dont move on until all threads are finished assigning to zero
    __syncthreads();

    //step size (bin size) should be 100/10 = 10
    float stepSize = MAX_RANDOM_NUMBER / (float)NUMBER_OF_BINS;

    //sort the values up until we run out of random numbers
    if (tid < NUMBER_OF_RANDOM_NUMBERS)
    {
		//make variable to temporarily hold current random number value
        float value = randomNumbers[tid];

        //use integer division to decide what bin a value falls into (/10)
        int bin = (int)(value / stepSize);

		//put case where the number generated is exactly 100 into the lat bin
        if (bin == NUMBER_OF_BINS)
		{
            bin = NUMBER_OF_BINS - 1;
		}

		//let user know if a value can't be sorted
		else if (bin > NUMBER_OF_BINS)
		{
			printf("\nan invalid value was generated: value = %f", value);
			return;
		}

        //use atomic add to add one to the bin the current value belongs to
        atomicAdd(&sharedHist[bin], 1);
    }

    //run until all values are finished sorting
    __syncthreads();

    //use first ten threads of each block and add their bin totals to the global bin totals
    if (threadIdx.x < NUMBER_OF_BINS)
    {
        atomicAdd(&hist[threadIdx.x], sharedHist[threadIdx.x]);
    }
}

int main()
{
	float time;
	timeval start, end;
	
	long int test = NUMBER_OF_RANDOM_NUMBERS;
	if(2147483647 < test)
	{
		printf("\nThe length of your vector is longer than the largest integer value allowed of 2,147,483,647.\n");
		printf("You should check your code.\n Good Bye\n");
		exit(0);
	}
	
	//Set the thread structure that you will be using on the GPU	
	SetUpCudaDevices();

	//Partitioning off the memory that you will be using and padding with zero vector will be a factor of block size.
	AllocateMemory();

	//Loading up values to be added.
	Innitialize();
	
	gettimeofday(&start, NULL);
	fillHistogramCPU();
	gettimeofday(&end, NULL);
	time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
	printf("\nTime on CPU = %.15f milliseconds\n", (time/1000.0));
	
	gettimeofday(&start, NULL);
	//Copy Memory from CPU to GPU		
	hipMemcpyAsync(RandomNumbersGPU, RandomNumbersCPU, NUMBER_OF_RANDOM_NUMBERS*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	fillHistogramGPU<<<GridSize,BlockSize>>>(RandomNumbersGPU, HistogramGPU);
	cudaErrorCheck(__FILE__, __LINE__);
	//Copy Memory from GPU to CPU	
	hipMemcpyAsync(HistogramCPUTemp, HistogramGPU, NUMBER_OF_BINS*sizeof(int), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	gettimeofday(&end, NULL);
	time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
	printf("\nTime on GPU = %.15f milliseconds\n", (time/1000.0));

	//CPU results
	for(int i = 0; i < NUMBER_OF_BINS; i++)
	{
		printf("\n CPU histogram bin %d is %d.", i, HistogramCPU[i]);
	}
	printf("\n");

	//GPU results
	for(int i = 0; i < NUMBER_OF_BINS; i++)
	{
		printf("\n GPU histogram bin %d is %d.", i, HistogramCPUTemp[i]);
	}
	printf("\n");
	
	//Check
	for(int i = 0; i < NUMBER_OF_BINS; i++)
	{
		printf("\n Deference in histogram bins %d is %d.", i, abs(HistogramCPUTemp[i] - HistogramCPU[i]));
	}
	
	//You're done so cleanup your mess.
	CleanUp();	
	
	printf("\n\n");
	return(0);
}
